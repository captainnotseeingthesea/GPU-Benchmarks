
#include "../FFT/parboil.h"
#include <stdio.h>
#include <stdlib.h>
#include "../benchmark_common.h"
#include "file.h"
#include "gpu_info.h"
#include "spmv_jds.h"
#include "jds_kernels.cu"
#include "convert_dataset.h"
#include <cutil_inline.h>


/*
static int generate_vector(float *x_vector, int dim) 
{	
	srand(54321);	
	for(int i=0;i<dim;i++)
	{
		x_vector[i] = (rand() / (float) RAND_MAX);
	}
	return 0;
}
*/

int main_SPMV(hipStream_t stream_app, pthread_mutex_t *mutexapp, bool flag) {
	struct pb_TimerSet timers;
	// struct pb_Parameters *parameters;
	
	
	
	
	
	printf("CUDA accelerated sparse matrix vector multiplication****\n");
	printf("Original version by Li-Wen Chang <lchang20@illinois.edu> and Shengzhao Wu<wu14@illinois.edu>\n");
	printf("This version maintained by Chris Rodrigues  ***********\n");
	// parameters = pb_ReadParameters(&argc, argv);
	// if ((parameters->inpFiles[0] == NULL) || (parameters->inpFiles[1] == NULL))
    // {
    //   fprintf(stderr, "Expecting two input filenames\n");
    //   exit(-1);
    // }

	
	pb_InitializeTimerSet(&timers);
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
	
	//parameters declaration
	int len;
	int depth;
	int dim;
	int pad=32;
	int nzcnt_len;
	
	//host memory allocation
	//matrix
	float *h_data;
	int *h_indices;
	int *h_ptr;
	int *h_perm;
	int *h_nzcnt;
	//vector
	float *h_Ax_vector;
    float *h_x_vector;
	
	//device memory allocation
	//matrix
	float *d_data;
	int *d_indices;
	int *d_ptr;
	int *d_perm;
	int *d_nzcnt;
	//vector
	float *d_Ax_vector;
    float *d_x_vector;
	
    //load matrix from files
	pb_SwitchToTimer(&timers, pb_TimerID_IO);
	//inputData(parameters->inpFiles[0], &len, &depth, &dim,&nzcnt_len,&pad,
	//    &h_data, &h_indices, &h_ptr,
	//    &h_perm, &h_nzcnt);
	int col_count;
	coo_to_jds(
		"SPMV/input/Dubcova3.mtx.bin", // bcsstk32.mtx, fidapm05.mtx, jgl009.mtx
		1, // row padding
		pad, // warp size, IMPORTANT: change in kernel as well
		1, // pack size
		1, // is mirrored?
		0, // binary matrix
		1, // debug level [0:2]
		&h_data, &h_ptr, &h_nzcnt, &h_indices, &h_perm,
		&col_count, &dim, &len, &nzcnt_len, &depth
	);
	

  h_Ax_vector=(float*)malloc(sizeof(float)*dim); 
  h_x_vector=(float*)malloc(sizeof(float)*dim);
  input_vec( "SPMV/input/vector.bin",h_x_vector,dim);

	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
	
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
	
	
	pb_SwitchToTimer(&timers, pb_TimerID_COPY);
	//memory allocation
	hipMalloc((void **)&d_data, len*sizeof(float));
	hipMalloc((void **)&d_indices, len*sizeof(int));
	hipMalloc((void **)&d_ptr, depth*sizeof(int));
	hipMalloc((void **)&d_perm, dim*sizeof(int));
	hipMalloc((void **)&d_nzcnt, nzcnt_len*sizeof(int));
	hipMalloc((void **)&d_x_vector, dim*sizeof(float));
	hipMalloc((void **)&d_Ax_vector,dim*sizeof(float));
	hipMemset( (void *) d_Ax_vector, 0, dim*sizeof(float));
	
	//memory copy
	hipMemcpyAsync(d_data, h_data, len*sizeof(float), hipMemcpyHostToDevice, stream_app);
	hipMemcpyAsync(d_indices, h_indices, len*sizeof(int), hipMemcpyHostToDevice, stream_app);
	hipMemcpyAsync(d_perm, h_perm, dim*sizeof(int), hipMemcpyHostToDevice, stream_app);
	hipMemcpyAsync(d_x_vector, h_x_vector, dim*sizeof(int), hipMemcpyHostToDevice, stream_app);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(jds_ptr_int), h_ptr, depth*sizeof(int), 0, hipMemcpyHostToDevice, stream_app);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(sh_zcnt_int), h_nzcnt,nzcnt_len*sizeof(int), 0, hipMemcpyHostToDevice, stream_app);
	
    if(flag)
		cutilSafeCall( hipStreamSynchronize(stream_app) );
	else
		cutilSafeCall( hipDeviceSynchronize() );
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
	unsigned int grid;
	unsigned int block;
    compute_active_thread(&block, &grid,nzcnt_len,pad, deviceProp.major,deviceProp.minor,
					deviceProp.warpSize,deviceProp.multiProcessorCount);

	
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(spmv_jds), hipFuncCachePreferL1);

	//main execution
	// pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);
	for (int i= 0; i<5; i++)
	{
		spmv_jds<<<grid, block, 0, stream_app>>>(d_Ax_vector, d_data, d_indices, d_perm, d_x_vector, d_nzcnt,dim);
		pthread_mutex_unlock (mutexapp);
		if(flag)
			cutilSafeCall( hipStreamSynchronize(stream_app) );
		else
			cutilSafeCall( hipDeviceSynchronize() );
		pthread_mutex_lock (mutexapp);
	}
	
							
    CUERR // check and clear any existing errors
	
    if(flag)
        cutilSafeCall( hipStreamSynchronize(stream_app) );
    else
        cutilSafeCall( hipDeviceSynchronize() );
	
	pb_SwitchToTimer(&timers, pb_TimerID_COPY);
	//HtoD memory copy
	hipMemcpyAsync(h_Ax_vector, d_Ax_vector,dim*sizeof(float), hipMemcpyDeviceToHost, stream_app);	

    if(flag)
        cutilSafeCall( hipStreamSynchronize(stream_app) );
    else
        cutilSafeCall( hipDeviceSynchronize() );

	hipFree(d_data);
    hipFree(d_indices);
    hipFree(d_ptr);
	hipFree(d_perm);
    hipFree(d_nzcnt);
    hipFree(d_x_vector);
	hipFree(d_Ax_vector);
 
	// if (parameters->outFile) {
		pb_SwitchToTimer(&timers, pb_TimerID_IO);
		outputData("SPMV/Dubcova3.mtx.out",h_Ax_vector,dim);
		
	// }
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
	
	free (h_data);
	free (h_indices);
	free (h_ptr);
	free (h_perm);
	free (h_nzcnt);
	free (h_Ax_vector);
	free (h_x_vector);
	pb_SwitchToTimer(&timers, pb_TimerID_NONE);

	pb_PrintTimerSet(&timers);
	// pb_FreeParameters(parameters);

	return 0;

}
