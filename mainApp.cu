#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <cutil_inline.h>
#include <unistd.h>
#include "benchmark_common.h"
#include <iomanip>

struct app_data
{
	long streamid;
	char *app_name;
	bool flag;
	bool mode;

};


hipStream_t *stream;
pthread_mutex_t mutexapp;

hipEvent_t app1_local_event;
hipEvent_t app2_local_event;
hipEvent_t app3_local_event;
hipEvent_t singleapp_local_event;

hipEvent_t app1_global_event;
hipEvent_t app2_global_event;
hipEvent_t app3_global_event;

hipEvent_t final_event;

int relaunch_count_1 = 1;
int relaunch_count_2 = 1;
int relaunch_count_3 = 1;

void callApp(char *app_name,hipStream_t stream_app, pthread_mutex_t *mutexapp, bool flag){
	if(strcmp(app_name,"VECTORADD")== 0)
		mainVectorAdd(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"VECTORADDONE")== 0)
		mainVectorAddOne(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"LUD")==0)
		main_LUD(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "HOTSPOT")==0)
		main_HOTSPOT(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "NW")== 0)
		main_NW(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "PATHFINDER")== 0)
		main_PATHFINDER(stream_app, mutexapp, flag); // has problems when running two apps concurrently
	else if(strcmp(app_name, "KMEANS")== 0) // can not run simultaneously due to synchronous symbol copy
	;
		// main_KMEANS(stream_app,mutexapp, flag);
	else if(strcmp(app_name, "SRADV1") == 0) // Has some problems
		main_SRADV1(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "SRADV2") == 0)
		main_SRADV2(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "BFS2") == 0)
		main_BFS2(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "NN") == 0)
		main_NN(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "SAD") == 0)
		main_SAD(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "3DS") == 0)
		main_threeDS(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "BLK") == 0)
		main_BlackScholes(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "BP") == 0)
		main_BP(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "CFD") == 0)
		main_cfd(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "CONS") == 0)
		main_CONS(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "FWT") == 0)
		main_FWT(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "FFT") == 0)
		main_fft(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "GUPS") == 0)
		main_gups(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "JPEG") == 0)
		main_JPEG(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "LPS") == 0)
		main_LPS(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "LIB") == 0)
		main_lib(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "LUH") == 0)
		main_lulesh(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "CN") == 0)
		main_CN(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "GRU") == 0)
		main_GRU(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "LSTM") == 0)
		main_LSTM(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"MUM") ==0)
		main_MUM(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"QTC") == 0)
		main_QTC(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"RED") ==0)
		main_RED(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"SC") ==0)
		main_streamcluster(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"SCAN") ==0)
		main_scan(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"SRAD") == 0)
		main_SRAD(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"TRD") ==0)
		main_TRD(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "BFS") == 0)
		main_BFS(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "MRI-Q") == 0)
		main_mri_q(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "SPMV") == 0)
		main_SPMV(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "ATAX") == 0)
		main_ATAX(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "BICG") == 0)
		main_BICG(stream_app, mutexapp, flag);
	else if(strcmp(app_name, "3DCONV") == 0)
		main_3DCONV(stream_app, mutexapp, flag);
}

void *app_1(void *app_arg) 
{
	pthread_mutex_lock (&mutexapp);
	struct app_data *my_data;
	
	my_data = (struct app_data *) app_arg; 
	bool appflag = my_data->flag;
    long mystreamid = my_data->streamid;

    cutilSafeCall(hipStreamCreate(&stream[mystreamid]));
    char *name = my_data->app_name;
	bool modeflag = my_data -> mode;
    start_app1:
    cutilSafeCall(hipEventCreate(&app1_local_event));
    
	callApp(name,stream[mystreamid], &mutexapp, appflag);

    cutilSafeCall(hipEventRecord(app1_local_event, stream[mystreamid]));
    cutilSafeCall(hipEventRecord(app1_global_event, stream[mystreamid]));
    
    loop_back_1:
	if ((!modeflag && (hipEventQuery(app1_local_event) == hipSuccess) && (hipEventQuery(app2_global_event) != hipSuccess)) || (modeflag && (hipEventQuery(app1_local_event) == hipSuccess) && (hipEventQuery(app2_global_event) != hipSuccess) && (hipEventQuery(app3_local_event) != hipSuccess))) {
		printf("I am relaunching App-1 (App-1 ran for %d times till now)\n", relaunch_count_1);
		relaunch_count_1++;
		hipEventDestroy(app1_local_event);
		goto start_app1;
	}
    
	if ((!modeflag && (hipEventQuery(app1_global_event) == hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess)) || (modeflag && (hipEventQuery(app1_global_event) == hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess) && (hipEventQuery(app3_global_event) == hipSuccess) ))  {
		cutilSafeCall(hipEventRecord(final_event, 0));
		
		
		printf("Both Applications completed once from app1\n");
		printf("App-1 ran for %d times\n", relaunch_count_1);
		printf("App-2 ran for %d times\n", relaunch_count_2);
		if(modeflag)
			printf("App-3 ran for %d times\n", relaunch_count_3);

		pthread_exit((void*) mystreamid);
		
		return NULL;
	}
    goto loop_back_1;
}

void *app_2(void *app_arg) 
{
	pthread_mutex_lock (&mutexapp);
	struct app_data *my_data;
	
	my_data = (struct app_data *) app_arg;
	
    bool appflag = my_data->flag;
    long mystreamid = my_data->streamid;

    cutilSafeCall(hipStreamCreate(&stream[mystreamid]));
    char *name = my_data->app_name;
	bool modeflag = my_data->mode;

    start_app2:
    cutilSafeCall(hipEventCreate(&app2_local_event));
    
	callApp(name,stream[mystreamid], &mutexapp, appflag);
    
    cutilSafeCall(hipEventRecord(app2_local_event, stream[mystreamid]));
    cutilSafeCall(hipEventRecord(app2_global_event, stream[mystreamid]));
    
    loop_back_2:
	if ((!modeflag && (hipEventQuery(app1_local_event) != hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess)) || (modeflag && (hipEventQuery(app1_local_event) != hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess) && (hipEventQuery(app3_local_event) != hipSuccess))) {
		printf("I am relaunching App-2 (App-2 ran for %d times till now)\n", relaunch_count_2);
		relaunch_count_2++;
		hipEventDestroy(app2_local_event);
		goto start_app2;
    }
	
    if ((!modeflag && (hipEventQuery(app1_global_event) == hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess)) || (modeflag && (hipEventQuery(app1_global_event) == hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess) && (hipEventQuery(app3_global_event) == hipSuccess)))  {
		
		cutilSafeCall(hipEventRecord(final_event, 0));
		
		printf("Both Applications completed once from app2\n");
		printf("App-1 ran for %d times\n", relaunch_count_1);
		printf("App-2 ran for %d times\n", relaunch_count_2);
		if(modeflag)
			printf("App-3 ran for %d times\n", relaunch_count_3);
		pthread_exit((void*) mystreamid);
		
		
		return NULL;
	}
    goto loop_back_2;
}

void *app_3(void *app_arg) 
{
	pthread_mutex_lock (&mutexapp);
	struct app_data *my_data;
	
	my_data = (struct app_data *) app_arg; 
	bool appflag = my_data->flag;
    long mystreamid = my_data->streamid;

    cutilSafeCall(hipStreamCreate(&stream[mystreamid]));
	printf("Hello from app3\n");
    char *name = my_data->app_name;
    start_app3:
    cutilSafeCall(hipEventCreate(&app3_local_event));
    
	
	callApp(name,stream[mystreamid], &mutexapp, appflag);


    cutilSafeCall(hipEventRecord(app3_local_event, stream[mystreamid]));
    cutilSafeCall(hipEventRecord(app3_global_event, stream[mystreamid]));
    
    loop_back_3:
	if ((hipEventQuery(app1_local_event) != hipSuccess) && (hipEventQuery(app2_global_event) != hipSuccess)&& (hipEventQuery(app3_global_event) != hipSuccess)) {
		printf("I am relaunching App-3 (App-3 ran for %d times till now)\n", relaunch_count_3);
		relaunch_count_3++;
		hipEventDestroy(app3_local_event);
		goto start_app3;
	}
    
	if ((hipEventQuery(app1_global_event) == hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess) && (hipEventQuery(app3_global_event) == hipSuccess))  {
		cutilSafeCall(hipEventRecord(final_event, 0));
		
		
		printf("Both Applications completed once from app1\n");
		printf("App-1 ran for %d times\n", relaunch_count_1);
		printf("App-2 ran for %d times\n", relaunch_count_2);
		printf("App-3 ran for %d times\n", relaunch_count_2);
		pthread_exit((void*) mystreamid);
		
		return NULL;
	}
    goto loop_back_3;
}

 void *single_app(void *app_arg){
    pthread_mutex_lock (&mutexapp);
	struct app_data *my_data;
	
	my_data = (struct app_data *) app_arg;
	
    bool appflag = my_data->flag;
    long mystreamid = my_data->streamid;

    cutilSafeCall(hipStreamCreate(&stream[mystreamid]));
    char *name = my_data->app_name;
    cutilSafeCall(hipEventCreate(&app1_local_event));
    
	callApp(name,stream[mystreamid], &mutexapp, appflag);  


	cutilSafeCall(hipEventRecord(final_event, 0));
	printf("Single app is completed\n");
	pthread_exit((void*) mystreamid);		
	return NULL;

 }
 void *single0_app(void *app_arg){
	pthread_mutex_lock (&mutexapp);
	struct app_data *my_data;
	
	my_data = (struct app_data *) app_arg; 
    long mystreamid = my_data->streamid;
	char *name = my_data->app_name;
	bool appflag = my_data->flag;
	callApp(name,0, &mutexapp, appflag);
	
	cutilSafeCall(hipEventRecord(final_event, 0));
	printf("Single app with 32 is completed\n");
	pthread_exit((void*) mystreamid);		
	return NULL;
 
 }

 
int main(int argc, char *argv[])
{

	typedef void* (*ptFunction)(void*);
	ptFunction functions[3];
	functions[0] = &app_1;
	functions[1]= &app_2;
	functions[2]= &app_3;
	void *status_single;
	pthread_t threads_single;
	struct app_data app_data_array_single;
	
	cutilSafeCall(hipEventCreate(&app1_global_event));
	cutilSafeCall(hipEventCreate(&app2_global_event));
	cutilSafeCall(hipEventCreate(&app3_global_event));
	cutilSafeCall(hipEventCreate(&final_event));
	pthread_mutex_init(&mutexapp, NULL);
	
    if(strcmp(argv[1],"-sing0")==0){

		printf("In main: creating thread %ld\n", 0);
		app_data_array_single.streamid = 0;
		app_data_array_single.app_name = argv[2];
		app_data_array_single.flag = false;
		app_data_array_single.mode = false;
		

	    (void) pthread_create(&threads_single, NULL,single0_app , (void *) &app_data_array_single);
		(void) pthread_join(threads_single, &status_single);
		printf("Main: completed join with thread %ld having a status of %ld\n",0,(long)status_single);
		
	}
	else if(strcmp(argv[1],"-sing") == 0){

		stream = (hipStream_t *)malloc(sizeof(hipStream_t));
		printf("In main: creating thread %ld\n", 0);
		app_data_array_single.streamid = 0;
		app_data_array_single.app_name = argv[2];
		app_data_array_single.flag = true;
		app_data_array_single.mode = false;
	    (void) pthread_create(&threads_single, NULL,single_app , (void *) &app_data_array_single);
		(void) pthread_join(threads_single, &status_single);
		printf("Main: completed join with thread %ld having a status of %ld\n",0,(long)status_single);
		hipStreamDestroy(stream[0]);
	}
	else if (strcmp(argv[1], "-apps")== 0){
		int num_thread=2;
	   	void *status[num_thread];
		struct app_data app_data_array[num_thread];
		pthread_t threads[num_thread];
		stream = (hipStream_t *)malloc(num_thread * sizeof(hipStream_t));
		for(long t=0; t<2; t++) {
			printf("In main: creating thread %ld\n", t);
			app_data_array[t].streamid = t;
			app_data_array[t].app_name = argv[2+t];
			app_data_array[t].flag = true;
			app_data_array[t].mode = false;
		}
	
	    (void) pthread_create(&threads[0], NULL, *functions[0], (void *) &app_data_array[0]);
		//sleep(1);
		(void) pthread_create(&threads[1], NULL, *functions[1], (void *) &app_data_array[1]);
		(void) pthread_join(threads[0], &status[0]);
		printf("Main: completed join with thread %ld having a status of %ld\n",0,(long)status[0]);
		(void) pthread_join(threads[1], &status[1]);
		printf("Main: completed join with thread %ld having a status of %ld\n",1,(long)status[1]);
		hipStreamDestroy(stream[0]);
		hipStreamDestroy(stream[1]);
	
	}
	else if (strcmp(argv[1], "-apps3")== 0){
		int num_thread=3;
	   	void *status[num_thread];
		struct app_data app_data_array[num_thread];
		pthread_t threads[num_thread];
		stream = (hipStream_t *)malloc(num_thread * sizeof(hipStream_t));
		for(long t=0; t<3; t++) {
			printf("In main: creating thread %ld\n", t);
			app_data_array[t].streamid = t;
			app_data_array[t].app_name = argv[2+t];
			app_data_array[t].flag = true;
			app_data_array[t].mode = true;
			
		}
	
	    (void) pthread_create(&threads[0], NULL, *functions[0], (void *) &app_data_array[0]);
		sleep(1);
		(void) pthread_create(&threads[1], NULL, *functions[1], (void *) &app_data_array[1]);
		sleep(1);
		(void) pthread_create(&threads[2], NULL, *functions[2], (void *) &app_data_array[2]);
		(void) pthread_join(threads[0], &status[0]);
		printf("Main: completed join with thread %ld having a status of %ld\n",0,(long)status[0]);
		(void) pthread_join(threads[1], &status[1]);
		printf("Main: completed join with thread %ld having a status of %ld\n",1,(long)status[1]);
		(void) pthread_join(threads[2], &status[2]);
		printf("Main: completed join with thread %ld having a status of %ld\n",2,(long)status[2]);
		hipStreamDestroy(stream[0]);
		hipStreamDestroy(stream[1]);
		hipStreamDestroy(stream[2]);
	
	}else if(strcmp(argv[1], "-list")== 0){
		printf("List of supported benchmarks\n");

		
	}
	else{
		printf("Usage: %s -apps app1 app2 OR Usage: %s -apps app1", argv[0],argv[0]);
		return 1;		
	}

	hipEventDestroy(app1_global_event);
	hipEventDestroy(app2_global_event);
	hipEventDestroy(app3_global_event);
	hipEventDestroy(final_event);

	pthread_mutex_destroy(&mutexapp);
	printf("Main: program completed. Exiting.\n");
		
	return 0;
}
