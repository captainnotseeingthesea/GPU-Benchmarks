/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/* 
 * C code for creating the Q data structure for fast convolution-based 
 * Hessian multiplication for arbitrary k-space trajectories.
 *
 * Inputs:
 * kx - VECTOR of kx values, same length as ky and kz
 * ky - VECTOR of ky values, same length as kx and kz
 * kz - VECTOR of kz values, same length as kx and ky
 * x  - VECTOR of x values, same length as y and z
 * y  - VECTOR of y values, same length as x and z
 * z  - VECTOR of z values, same length as x and y
 * phi - VECTOR of the Fourier transform of the spatial basis 
 *      function, evaluated at [kx, ky, kz].  Same length as kx, ky, and kz.
 *
 * recommended g++ options:
 *  -O3 -lm -ffast-math -funroll-all-loops
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include <malloc.h>
#include <cutil_inline.h>
#include "../FFT/parboil.h"
#include "../benchmark_common.h"
#include "file.h"
#include "computeQ.cu"

static void
setupMemoryGPU(int num, int size, float*& dev_ptr, float*& host_ptr, hipStream_t stream_app)
{
  hipMalloc ((void **) &dev_ptr, num * size);
  CUDA_ERRCK;
  hipMemcpyAsync (dev_ptr, host_ptr, num * size, hipMemcpyHostToDevice, stream_app);
  CUDA_ERRCK;
}

static void
cleanupMemoryGPU(int num, int size, float *& dev_ptr, float * host_ptr, hipStream_t stream_app)
{
  hipMemcpyAsync (host_ptr, dev_ptr, num * size, hipMemcpyDeviceToHost, stream_app);
  CUDA_ERRCK;
  hipFree(dev_ptr);
  CUDA_ERRCK;
}

int
main_mri_q (hipStream_t stream_app, pthread_mutex_t *mutexapp, bool flag) {
  int numX, numK;		/* Number of X and K values */
  int original_numK;		/* Number of K values in input file */
  float *kx, *ky, *kz;		/* K trajectory (3D vectors) */
  float *x, *y, *z;		/* X coordinates (3D vectors) */
  float *phiR, *phiI;		/* Phi values (complex) */
  float *phiMag;		/* Magnitude of Phi */
  float *Qr, *Qi;		/* Q signal (complex) */

  struct kValues* kVals;

  // struct pb_Parameters *params;
  struct pb_TimerSet timers;

  pb_InitializeTimerSet(&timers);

  /* Read command line */
  // params = pb_ReadParameters(&argc, argv);
  // if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL))
  //   {
  //     fprintf(stderr, "Expecting one input filename\n");
  //     exit(-1);
  //   }
  
  /* Read in data */
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  inputData("./MRI-Q/input/32_32_32_dataset.bin",
	    &original_numK, &numX,
	    &kx, &ky, &kz,
	    &x, &y, &z,
	    &phiR, &phiI);

  /* Reduce the number of k-space samples if a number is given
   * on the command line */
  // if (argc < 2)
    numK = original_numK;
  // else
  //   {
  //     int inputK;
  //     char *end;
  //     inputK = strtol(argv[1], &end, 10);
  //     if (end == argv[1])
	// {
	//   fprintf(stderr, "Expecting an integer parameter\n");
	//   exit(-1);
	// }

  //     numK = MIN(inputK, original_numK);
  //   }

  printf("%d pixels in output; %d samples in trajectory; using %d samples\n",
         numX, original_numK, numK);

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  /* Create CPU data structures */
  createDataStructsCPU(numK, numX, &phiMag, &Qr, &Qi);

  /* GPU section 1 (precompute PhiMag) */
  {
    /* Mirror several data structures on the device */
    float *phiR_d, *phiI_d;
    float *phiMag_d;

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);
    setupMemoryGPU(numK, sizeof(float), phiR_d, phiR, stream_app);
    setupMemoryGPU(numK, sizeof(float), phiI_d, phiI, stream_app);
    hipMalloc((void **)&phiMag_d, numK * sizeof(float));
    CUDA_ERRCK;

    if(flag)
      cutilSafeCall( hipStreamSynchronize(stream_app) );
    else
      cutilSafeCall( hipDeviceSynchronize() );
    // pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

    computePhiMag_GPU(numK, phiR_d, phiI_d, phiMag_d, stream_app, mutexapp, flag);

    if(flag)
        cutilSafeCall( hipStreamSynchronize(stream_app) );
    else
        cutilSafeCall( hipDeviceSynchronize() );
    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    cleanupMemoryGPU(numK, sizeof(float), phiMag_d, phiMag, stream_app);
    hipFree(phiR_d);
    hipFree(phiI_d);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  kVals = (struct kValues*)calloc(numK, sizeof (struct kValues));
  for (int k = 0; k < numK; k++) {
    kVals[k].Kx = kx[k];
    kVals[k].Ky = ky[k];
    kVals[k].Kz = kz[k];
    kVals[k].PhiMag = phiMag[k];
  }

  free(phiMag);

  /* GPU section 2 */
  {
    float *x_d, *y_d, *z_d;
    float *Qr_d, *Qi_d;

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    setupMemoryGPU(numX, sizeof(float), x_d, x, stream_app);
    setupMemoryGPU(numX, sizeof(float), y_d, y, stream_app);
    setupMemoryGPU(numX, sizeof(float), z_d, z, stream_app);
    hipMalloc((void **)&Qr_d, numX * sizeof(float));
    CUDA_ERRCK;
    hipMemset((void *)Qr_d, 0, numX * sizeof(float));
    hipMalloc((void **)&Qi_d, numX * sizeof(float));
    CUDA_ERRCK;
    hipMemset((void *)Qi_d, 0, numX * sizeof(float));

    if(flag)
        cutilSafeCall( hipStreamSynchronize(stream_app) );
    else
        cutilSafeCall( hipDeviceSynchronize() );
    // pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

    computeQ_GPU(numK, numX, x_d, y_d, z_d, kVals, Qr_d, Qi_d, stream_app, mutexapp, flag);

    if(flag)
        cutilSafeCall( hipStreamSynchronize(stream_app) );
    else
        cutilSafeCall( hipDeviceSynchronize() );
    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    cleanupMemoryGPU(numX, sizeof(float), Qr_d, Qr, stream_app);
    cleanupMemoryGPU(numX, sizeof(float), Qi_d, Qi, stream_app);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  // if (params->outFile)
  //   {
  //     /* Write Q to file */
      pb_SwitchToTimer(&timers, pb_TimerID_IO);
      outputData("./MRI-Q/32_32_32_dataset.out", Qr, Qi, numX);
      pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
  //   }

  free (kx);
  free (ky);
  free (kz);
  free (x);
  free (y);
  free (z);
  free (phiR);
  free (phiI);
  free (kVals);
  free (Qr);
  free (Qi);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);

  // pb_FreeParameters(params);

  return 0;
}
